#include "hip/hip_runtime.h"
/**
 * AES.cpp
 *
 * The Advanced Encryption Standard (AES, aka AES) block cipher,
 * designed by J. Daemen and V. Rijmen.
 *
 * @author Paulo S. L. M. Barreto, Simon Waloschek, Benedikt Krueger
 *
 * This software is hereby placed in the public domain.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHORS ''AS IS'' AND ANY EXPRESS
 * OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHORS OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
 * BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
 * WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
 * OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <assert.h>
#include <string.h>
#include <stdlib.h>

#ifdef BENCHMARK
#include <stdio.h>
#include <time.h>
#endif

#include "AES.h"
#include "AES.tab"

#define FULL_UNROLL

#ifdef _MSC_VER
#define SWAP(x) (_lrotl(x, 8) & 0x00ff00ff | _lrotr(x, 8) & 0xff00ff00)
#define GETWORD(p) SWAP(*((uint *)(p)))
#define PUTWORD(ct, st) (*((uint *)(ct)) = SWAP((st)))
#else
#define GETWORD(pt) (((uint)(pt)[0] << 24) ^ ((uint)(pt)[1] << 16) ^ ((uint)(pt)[2] <<  8) ^ ((uint)(pt)[3]))
#define PUTWORD(ct, st) ((ct)[0] = (byte)((st) >> 24), (ct)[1] = (byte)((st) >> 16), (ct)[2] = (byte)((st) >>  8), (ct)[3] = (byte)(st), (st))
#endif

//////////////////////////////////////////////////////////////////////
// Construction/Destruction
//////////////////////////////////////////////////////////////////////

AES::AES() {
    hipMalloc((void**)&ce_sched, sizeof(e_sched));
    hipMalloc((void**)&cd_sched, sizeof(d_sched));
}

AES::~AES() {
    Nr = 0;
    memset(e_sched, 0, sizeof(e_sched));
    memset(d_sched, 0, sizeof(d_sched));

    hipFree(ce_sched);
    hipFree(cd_sched);
}

//////////////////////////////////////////////////////////////////////
// Support methods
//////////////////////////////////////////////////////////////////////

void AES::ExpandKey(const byte *cipherKey, uint keyBits) {
    uint *rek = e_sched;
    uint i = 0;
    uint temp;
    rek[0] = GETWORD(cipherKey     );
    rek[1] = GETWORD(cipherKey +  4);
    rek[2] = GETWORD(cipherKey +  8);
    rek[3] = GETWORD(cipherKey + 12);
    if (keyBits == 128) {
        for (;;) {
            temp  = rek[3];
            rek[4] = rek[0] ^
                (Te4[(temp >> 16) & 0xff] & 0xff000000) ^
                (Te4[(temp >>  8) & 0xff] & 0x00ff0000) ^
                (Te4[(temp      ) & 0xff] & 0x0000ff00) ^
                (Te4[(temp >> 24)       ] & 0x000000ff) ^
                rcon[i];
            rek[5] = rek[1] ^ rek[4];
            rek[6] = rek[2] ^ rek[5];
            rek[7] = rek[3] ^ rek[6];
            if (++i == 10) {
                Nr = 10;
                return;
            }
            rek += 4;
        }
    }
    rek[4] = GETWORD(cipherKey + 16);
    rek[5] = GETWORD(cipherKey + 20);
    if (keyBits == 192) {
        for (;;) {
            temp = rek[ 5];
            rek[ 6] = rek[ 0] ^
                (Te4[(temp >> 16) & 0xff] & 0xff000000) ^
                (Te4[(temp >>  8) & 0xff] & 0x00ff0000) ^
                (Te4[(temp      ) & 0xff] & 0x0000ff00) ^
                (Te4[(temp >> 24)       ] & 0x000000ff) ^
                rcon[i];
            rek[ 7] = rek[ 1] ^ rek[ 6];
            rek[ 8] = rek[ 2] ^ rek[ 7];
            rek[ 9] = rek[ 3] ^ rek[ 8];
            if (++i == 8) {
                Nr = 12;
                return;
            }
            rek[10] = rek[ 4] ^ rek[ 9];
            rek[11] = rek[ 5] ^ rek[10];
            rek += 6;
        }
    }
    rek[6] = GETWORD(cipherKey + 24);
    rek[7] = GETWORD(cipherKey + 28);
    if (keyBits == 256) {
        for (;;) {
            temp = rek[ 7];
            rek[ 8] = rek[ 0] ^
                (Te4[(temp >> 16) & 0xff] & 0xff000000) ^
                (Te4[(temp >>  8) & 0xff] & 0x00ff0000) ^
                (Te4[(temp      ) & 0xff] & 0x0000ff00) ^
                (Te4[(temp >> 24)       ] & 0x000000ff) ^
                rcon[i];
            rek[ 9] = rek[ 1] ^ rek[ 8];
            rek[10] = rek[ 2] ^ rek[ 9];
            rek[11] = rek[ 3] ^ rek[10];
            if (++i == 7) {
                Nr = 14;
                return;
            }
            temp = rek[11];
            rek[12] = rek[ 4] ^
                (Te4[(temp >> 24)       ] & 0xff000000) ^
                (Te4[(temp >> 16) & 0xff] & 0x00ff0000) ^
                (Te4[(temp >>  8) & 0xff] & 0x0000ff00) ^
                (Te4[(temp      ) & 0xff] & 0x000000ff);
            rek[13] = rek[ 5] ^ rek[12];
            rek[14] = rek[ 6] ^ rek[13];
            rek[15] = rek[ 7] ^ rek[14];
            rek += 8;
        }
    }
    Nr = 0; // this should never happen
}

void AES::InvertKey() {
    uint *rek = e_sched;
    uint *rdk = d_sched;
    assert(Nr == 10 || Nr == 12 || Nr == 14);
    rek += 4*Nr;
    /* apply the inverse MixColumn transform to all round keys but the first and the last: */
    memcpy(rdk, rek, 16);
    rdk += 4;
    rek -= 4;
    for (uint r = 1; r < Nr; r++) {
        rdk[0] =
            Td0[Te4[(rek[0] >> 24)       ] & 0xff] ^
            Td1[Te4[(rek[0] >> 16) & 0xff] & 0xff] ^
            Td2[Te4[(rek[0] >>  8) & 0xff] & 0xff] ^
            Td3[Te4[(rek[0]      ) & 0xff] & 0xff];
        rdk[1] =
            Td0[Te4[(rek[1] >> 24)       ] & 0xff] ^
            Td1[Te4[(rek[1] >> 16) & 0xff] & 0xff] ^
            Td2[Te4[(rek[1] >>  8) & 0xff] & 0xff] ^
            Td3[Te4[(rek[1]      ) & 0xff] & 0xff];
        rdk[2] =
            Td0[Te4[(rek[2] >> 24)       ] & 0xff] ^
            Td1[Te4[(rek[2] >> 16) & 0xff] & 0xff] ^
            Td2[Te4[(rek[2] >>  8) & 0xff] & 0xff] ^
            Td3[Te4[(rek[2]      ) & 0xff] & 0xff];
        rdk[3] =
            Td0[Te4[(rek[3] >> 24)       ] & 0xff] ^
            Td1[Te4[(rek[3] >> 16) & 0xff] & 0xff] ^
            Td2[Te4[(rek[3] >>  8) & 0xff] & 0xff] ^
            Td3[Te4[(rek[3]      ) & 0xff] & 0xff];
        rdk += 4;
        rek -= 4;
    }
    memcpy(rdk, rek, 16);
}

//////////////////////////////////////////////////////////////////////
// Public Interface
//////////////////////////////////////////////////////////////////////

void AES::byte2int(const byte *b, uint *i) {
    i[0] = GETWORD(b     );
    i[1] = GETWORD(b +  4);
    i[2] = GETWORD(b +  8);
    i[3] = GETWORD(b + 12);
}

void AES::int2byte(const uint *i, byte *b) {
    PUTWORD(b     , i[0]);
    PUTWORD(b +  4, i[1]);
    PUTWORD(b +  8, i[2]);
    PUTWORD(b + 12, i[3]);
}

void AES::makeKey(const byte *cipherKey, uint keySize, uint dir) {
    switch (keySize) {
    case 16:
    case 24:
    case 32:
        keySize <<= 3;
        break;
    case 128:
    case 192:
    case 256:
        break;
    default:
        throw "Invalid AES key size";
    }
    assert(dir <= DIR_BOTH);
    if (dir != DIR_NONE) {
        ExpandKey(cipherKey, keySize);
        hipMemcpy(ce_sched, e_sched, sizeof(e_sched), hipMemcpyHostToDevice);
        if (dir & DIR_DECRYPT) {
            InvertKey();
            hipMemcpy(cd_sched, d_sched, sizeof(e_sched), hipMemcpyHostToDevice);
        }
    }
}

void AES::encrypt(const uint *pt, uint *ct, uint n = 1) {
	uint *cpt, *cct;
	uint size = (n << 2)*sizeof(uint);

	hipMalloc((void**)&cpt, size);
	hipMalloc((void**)&cct, size);
	hipMemcpy(cpt, pt, size, hipMemcpyHostToDevice);

    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

	uint blocks, threads = 1;
	if(n != 1) {
		threads = (n < prop.maxThreadsPerBlock*2) ? n / 2 : prop.maxThreadsPerBlock;
	}
	blocks = n / threads;

	dim3 dimBlock(threads);
	dim3 dimGrid(blocks);

	AES_encrypt<<<dimGrid, dimBlock, size>>>(cpt, cct, ce_sched, Nr);

	hipMemcpy(ct, cct, size, hipMemcpyDeviceToHost);
	hipFree(cpt);
	hipFree(cct);
}

void AES::decrypt(const uint *ct, uint *pt, uint n = 1) {
	uint *cpt, *cct;
	uint size = (n << 2)*sizeof(uint);

	hipMalloc((void**)&cpt, size);
	hipMalloc((void**)&cct, size);
	hipMemcpy(cct, ct, size, hipMemcpyHostToDevice);

    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

	uint blocks, threads = 1;
	if(n != 1) {
		threads = (n < prop.maxThreadsPerBlock*2) ? n / 2 : prop.maxThreadsPerBlock;
	}
	blocks = n / threads;

	dim3 dimBlock(threads);
	dim3 dimGrid(blocks);

	AES_decrypt<<<dimGrid, dimBlock, size>>>(cct, cpt, cd_sched, Nr);

	hipMemcpy(pt, cpt, size, hipMemcpyDeviceToHost);
	hipFree(cpt);
	hipFree(cct);
}

__global__ void AES_encrypt(const uint *pt, uint *ct, uint *rek, uint Nr) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int i = x + y * gridDim.x * blockDim.x;
    int offset = i << 2;

    __shared__ __device__ uint s0, s1, s2, s3, t0, t1, t2, t3;

    s0 = pt[offset + 0] ^ rek[0];
    s1 = pt[offset + 1] ^ rek[1];
    s2 = pt[offset + 2] ^ rek[2];
    s3 = pt[offset + 3] ^ rek[3];

    /* round 1: */
    t0 = cTe0[s0 >> 24] ^ cTe1[(s1 >> 16) & 0xff] ^ cTe2[(s2 >>  8) & 0xff] ^ cTe3[s3 & 0xff] ^ rek[ 4];
    t1 = cTe0[s1 >> 24] ^ cTe1[(s2 >> 16) & 0xff] ^ cTe2[(s3 >>  8) & 0xff] ^ cTe3[s0 & 0xff] ^ rek[ 5];
    t2 = cTe0[s2 >> 24] ^ cTe1[(s3 >> 16) & 0xff] ^ cTe2[(s0 >>  8) & 0xff] ^ cTe3[s1 & 0xff] ^ rek[ 6];
    t3 = cTe0[s3 >> 24] ^ cTe1[(s0 >> 16) & 0xff] ^ cTe2[(s1 >>  8) & 0xff] ^ cTe3[s2 & 0xff] ^ rek[ 7];
    /* round 2: */
    s0 = cTe0[t0 >> 24] ^ cTe1[(t1 >> 16) & 0xff] ^ cTe2[(t2 >>  8) & 0xff] ^ cTe3[t3 & 0xff] ^ rek[ 8];
    s1 = cTe0[t1 >> 24] ^ cTe1[(t2 >> 16) & 0xff] ^ cTe2[(t3 >>  8) & 0xff] ^ cTe3[t0 & 0xff] ^ rek[ 9];
    s2 = cTe0[t2 >> 24] ^ cTe1[(t3 >> 16) & 0xff] ^ cTe2[(t0 >>  8) & 0xff] ^ cTe3[t1 & 0xff] ^ rek[10];
    s3 = cTe0[t3 >> 24] ^ cTe1[(t0 >> 16) & 0xff] ^ cTe2[(t1 >>  8) & 0xff] ^ cTe3[t2 & 0xff] ^ rek[11];
    /* round 3: */
    t0 = cTe0[s0 >> 24] ^ cTe1[(s1 >> 16) & 0xff] ^ cTe2[(s2 >>  8) & 0xff] ^ cTe3[s3 & 0xff] ^ rek[12];
    t1 = cTe0[s1 >> 24] ^ cTe1[(s2 >> 16) & 0xff] ^ cTe2[(s3 >>  8) & 0xff] ^ cTe3[s0 & 0xff] ^ rek[13];
    t2 = cTe0[s2 >> 24] ^ cTe1[(s3 >> 16) & 0xff] ^ cTe2[(s0 >>  8) & 0xff] ^ cTe3[s1 & 0xff] ^ rek[14];
    t3 = cTe0[s3 >> 24] ^ cTe1[(s0 >> 16) & 0xff] ^ cTe2[(s1 >>  8) & 0xff] ^ cTe3[s2 & 0xff] ^ rek[15];
    /* round 4: */
    s0 = cTe0[t0 >> 24] ^ cTe1[(t1 >> 16) & 0xff] ^ cTe2[(t2 >>  8) & 0xff] ^ cTe3[t3 & 0xff] ^ rek[16];
    s1 = cTe0[t1 >> 24] ^ cTe1[(t2 >> 16) & 0xff] ^ cTe2[(t3 >>  8) & 0xff] ^ cTe3[t0 & 0xff] ^ rek[17];
    s2 = cTe0[t2 >> 24] ^ cTe1[(t3 >> 16) & 0xff] ^ cTe2[(t0 >>  8) & 0xff] ^ cTe3[t1 & 0xff] ^ rek[18];
    s3 = cTe0[t3 >> 24] ^ cTe1[(t0 >> 16) & 0xff] ^ cTe2[(t1 >>  8) & 0xff] ^ cTe3[t2 & 0xff] ^ rek[19];
    /* round 5: */
    t0 = cTe0[s0 >> 24] ^ cTe1[(s1 >> 16) & 0xff] ^ cTe2[(s2 >>  8) & 0xff] ^ cTe3[s3 & 0xff] ^ rek[20];
    t1 = cTe0[s1 >> 24] ^ cTe1[(s2 >> 16) & 0xff] ^ cTe2[(s3 >>  8) & 0xff] ^ cTe3[s0 & 0xff] ^ rek[21];
    t2 = cTe0[s2 >> 24] ^ cTe1[(s3 >> 16) & 0xff] ^ cTe2[(s0 >>  8) & 0xff] ^ cTe3[s1 & 0xff] ^ rek[22];
    t3 = cTe0[s3 >> 24] ^ cTe1[(s0 >> 16) & 0xff] ^ cTe2[(s1 >>  8) & 0xff] ^ cTe3[s2 & 0xff] ^ rek[23];
    /* round 6: */
    s0 = cTe0[t0 >> 24] ^ cTe1[(t1 >> 16) & 0xff] ^ cTe2[(t2 >>  8) & 0xff] ^ cTe3[t3 & 0xff] ^ rek[24];
    s1 = cTe0[t1 >> 24] ^ cTe1[(t2 >> 16) & 0xff] ^ cTe2[(t3 >>  8) & 0xff] ^ cTe3[t0 & 0xff] ^ rek[25];
    s2 = cTe0[t2 >> 24] ^ cTe1[(t3 >> 16) & 0xff] ^ cTe2[(t0 >>  8) & 0xff] ^ cTe3[t1 & 0xff] ^ rek[26];
    s3 = cTe0[t3 >> 24] ^ cTe1[(t0 >> 16) & 0xff] ^ cTe2[(t1 >>  8) & 0xff] ^ cTe3[t2 & 0xff] ^ rek[27];
    /* round 7: */
    t0 = cTe0[s0 >> 24] ^ cTe1[(s1 >> 16) & 0xff] ^ cTe2[(s2 >>  8) & 0xff] ^ cTe3[s3 & 0xff] ^ rek[28];
    t1 = cTe0[s1 >> 24] ^ cTe1[(s2 >> 16) & 0xff] ^ cTe2[(s3 >>  8) & 0xff] ^ cTe3[s0 & 0xff] ^ rek[29];
    t2 = cTe0[s2 >> 24] ^ cTe1[(s3 >> 16) & 0xff] ^ cTe2[(s0 >>  8) & 0xff] ^ cTe3[s1 & 0xff] ^ rek[30];
    t3 = cTe0[s3 >> 24] ^ cTe1[(s0 >> 16) & 0xff] ^ cTe2[(s1 >>  8) & 0xff] ^ cTe3[s2 & 0xff] ^ rek[31];
    /* round 8: */
    s0 = cTe0[t0 >> 24] ^ cTe1[(t1 >> 16) & 0xff] ^ cTe2[(t2 >>  8) & 0xff] ^ cTe3[t3 & 0xff] ^ rek[32];
    s1 = cTe0[t1 >> 24] ^ cTe1[(t2 >> 16) & 0xff] ^ cTe2[(t3 >>  8) & 0xff] ^ cTe3[t0 & 0xff] ^ rek[33];
    s2 = cTe0[t2 >> 24] ^ cTe1[(t3 >> 16) & 0xff] ^ cTe2[(t0 >>  8) & 0xff] ^ cTe3[t1 & 0xff] ^ rek[34];
    s3 = cTe0[t3 >> 24] ^ cTe1[(t0 >> 16) & 0xff] ^ cTe2[(t1 >>  8) & 0xff] ^ cTe3[t2 & 0xff] ^ rek[35];
    /* round 9: */
    t0 = cTe0[s0 >> 24] ^ cTe1[(s1 >> 16) & 0xff] ^ cTe2[(s2 >>  8) & 0xff] ^ cTe3[s3 & 0xff] ^ rek[36];
    t1 = cTe0[s1 >> 24] ^ cTe1[(s2 >> 16) & 0xff] ^ cTe2[(s3 >>  8) & 0xff] ^ cTe3[s0 & 0xff] ^ rek[37];
    t2 = cTe0[s2 >> 24] ^ cTe1[(s3 >> 16) & 0xff] ^ cTe2[(s0 >>  8) & 0xff] ^ cTe3[s1 & 0xff] ^ rek[38];
    t3 = cTe0[s3 >> 24] ^ cTe1[(s0 >> 16) & 0xff] ^ cTe2[(s1 >>  8) & 0xff] ^ cTe3[s2 & 0xff] ^ rek[39];
    if (Nr > 10) {
        /* round 10: */
        s0 = cTe0[t0 >> 24] ^ cTe1[(t1 >> 16) & 0xff] ^ cTe2[(t2 >>  8) & 0xff] ^ cTe3[t3 & 0xff] ^ rek[40];
        s1 = cTe0[t1 >> 24] ^ cTe1[(t2 >> 16) & 0xff] ^ cTe2[(t3 >>  8) & 0xff] ^ cTe3[t0 & 0xff] ^ rek[41];
        s2 = cTe0[t2 >> 24] ^ cTe1[(t3 >> 16) & 0xff] ^ cTe2[(t0 >>  8) & 0xff] ^ cTe3[t1 & 0xff] ^ rek[42];
        s3 = cTe0[t3 >> 24] ^ cTe1[(t0 >> 16) & 0xff] ^ cTe2[(t1 >>  8) & 0xff] ^ cTe3[t2 & 0xff] ^ rek[43];
        /* round 11: */
        t0 = cTe0[s0 >> 24] ^ cTe1[(s1 >> 16) & 0xff] ^ cTe2[(s2 >>  8) & 0xff] ^ cTe3[s3 & 0xff] ^ rek[44];
        t1 = cTe0[s1 >> 24] ^ cTe1[(s2 >> 16) & 0xff] ^ cTe2[(s3 >>  8) & 0xff] ^ cTe3[s0 & 0xff] ^ rek[45];
        t2 = cTe0[s2 >> 24] ^ cTe1[(s3 >> 16) & 0xff] ^ cTe2[(s0 >>  8) & 0xff] ^ cTe3[s1 & 0xff] ^ rek[46];
        t3 = cTe0[s3 >> 24] ^ cTe1[(s0 >> 16) & 0xff] ^ cTe2[(s1 >>  8) & 0xff] ^ cTe3[s2 & 0xff] ^ rek[47];
        if (Nr > 12) {
            /* round 12: */
            s0 = cTe0[t0 >> 24] ^ cTe1[(t1 >> 16) & 0xff] ^ cTe2[(t2 >>  8) & 0xff] ^ cTe3[t3 & 0xff] ^ rek[48];
            s1 = cTe0[t1 >> 24] ^ cTe1[(t2 >> 16) & 0xff] ^ cTe2[(t3 >>  8) & 0xff] ^ cTe3[t0 & 0xff] ^ rek[49];
            s2 = cTe0[t2 >> 24] ^ cTe1[(t3 >> 16) & 0xff] ^ cTe2[(t0 >>  8) & 0xff] ^ cTe3[t1 & 0xff] ^ rek[50];
            s3 = cTe0[t3 >> 24] ^ cTe1[(t0 >> 16) & 0xff] ^ cTe2[(t1 >>  8) & 0xff] ^ cTe3[t2 & 0xff] ^ rek[51];
            /* round 13: */
            t0 = cTe0[s0 >> 24] ^ cTe1[(s1 >> 16) & 0xff] ^ cTe2[(s2 >>  8) & 0xff] ^ cTe3[s3 & 0xff] ^ rek[52];
            t1 = cTe0[s1 >> 24] ^ cTe1[(s2 >> 16) & 0xff] ^ cTe2[(s3 >>  8) & 0xff] ^ cTe3[s0 & 0xff] ^ rek[53];
            t2 = cTe0[s2 >> 24] ^ cTe1[(s3 >> 16) & 0xff] ^ cTe2[(s0 >>  8) & 0xff] ^ cTe3[s1 & 0xff] ^ rek[54];
            t3 = cTe0[s3 >> 24] ^ cTe1[(s0 >> 16) & 0xff] ^ cTe2[(s1 >>  8) & 0xff] ^ cTe3[s2 & 0xff] ^ rek[55];
        }
    }
    rek += Nr << 2;

    ct[offset + 0] =
        (cTe4[(t0 >> 24)       ] & 0xff000000) ^
        (cTe4[(t1 >> 16) & 0xff] & 0x00ff0000) ^
        (cTe4[(t2 >>  8) & 0xff] & 0x0000ff00) ^
        (cTe4[(t3      ) & 0xff] & 0x000000ff) ^
        rek[0];
    ct[offset + 1] =
        (cTe4[(t1 >> 24)       ] & 0xff000000) ^
        (cTe4[(t2 >> 16) & 0xff] & 0x00ff0000) ^
        (cTe4[(t3 >>  8) & 0xff] & 0x0000ff00) ^
        (cTe4[(t0      ) & 0xff] & 0x000000ff) ^
        rek[1];
    ct[offset + 2] =
        (cTe4[(t2 >> 24)       ] & 0xff000000) ^
        (cTe4[(t3 >> 16) & 0xff] & 0x00ff0000) ^
        (cTe4[(t0 >>  8) & 0xff] & 0x0000ff00) ^
        (cTe4[(t1      ) & 0xff] & 0x000000ff) ^
        rek[2];
    ct[offset + 3] =
        (cTe4[(t3 >> 24)       ] & 0xff000000) ^
        (cTe4[(t0 >> 16) & 0xff] & 0x00ff0000) ^
        (cTe4[(t1 >>  8) & 0xff] & 0x0000ff00) ^
        (cTe4[(t2      ) & 0xff] & 0x000000ff) ^
        rek[3];
}

__global__ void AES_decrypt(const uint *ct, uint *pt, uint *rdk, uint Nr) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int i = x + y * gridDim.x * blockDim.x;
    int offset = i << 2;

    __shared__ __device__ uint s0, s1, s2, s3, t0, t1, t2, t3;

    s0 = ct[offset + 0] ^ rdk[0];
    s1 = ct[offset + 1] ^ rdk[1];
    s2 = ct[offset + 2] ^ rdk[2];
    s3 = ct[offset + 3] ^ rdk[3];

    /* round 1: */
    t0 = cTd0[s0 >> 24] ^ cTd1[(s3 >> 16) & 0xff] ^ cTd2[(s2 >>  8) & 0xff] ^ cTd3[s1 & 0xff] ^ rdk[ 4];
    t1 = cTd0[s1 >> 24] ^ cTd1[(s0 >> 16) & 0xff] ^ cTd2[(s3 >>  8) & 0xff] ^ cTd3[s2 & 0xff] ^ rdk[ 5];
    t2 = cTd0[s2 >> 24] ^ cTd1[(s1 >> 16) & 0xff] ^ cTd2[(s0 >>  8) & 0xff] ^ cTd3[s3 & 0xff] ^ rdk[ 6];
    t3 = cTd0[s3 >> 24] ^ cTd1[(s2 >> 16) & 0xff] ^ cTd2[(s1 >>  8) & 0xff] ^ cTd3[s0 & 0xff] ^ rdk[ 7];
    /* round 2: */
    s0 = cTd0[t0 >> 24] ^ cTd1[(t3 >> 16) & 0xff] ^ cTd2[(t2 >>  8) & 0xff] ^ cTd3[t1 & 0xff] ^ rdk[ 8];
    s1 = cTd0[t1 >> 24] ^ cTd1[(t0 >> 16) & 0xff] ^ cTd2[(t3 >>  8) & 0xff] ^ cTd3[t2 & 0xff] ^ rdk[ 9];
    s2 = cTd0[t2 >> 24] ^ cTd1[(t1 >> 16) & 0xff] ^ cTd2[(t0 >>  8) & 0xff] ^ cTd3[t3 & 0xff] ^ rdk[10];
    s3 = cTd0[t3 >> 24] ^ cTd1[(t2 >> 16) & 0xff] ^ cTd2[(t1 >>  8) & 0xff] ^ cTd3[t0 & 0xff] ^ rdk[11];
    /* round 3: */
    t0 = cTd0[s0 >> 24] ^ cTd1[(s3 >> 16) & 0xff] ^ cTd2[(s2 >>  8) & 0xff] ^ cTd3[s1 & 0xff] ^ rdk[12];
    t1 = cTd0[s1 >> 24] ^ cTd1[(s0 >> 16) & 0xff] ^ cTd2[(s3 >>  8) & 0xff] ^ cTd3[s2 & 0xff] ^ rdk[13];
    t2 = cTd0[s2 >> 24] ^ cTd1[(s1 >> 16) & 0xff] ^ cTd2[(s0 >>  8) & 0xff] ^ cTd3[s3 & 0xff] ^ rdk[14];
    t3 = cTd0[s3 >> 24] ^ cTd1[(s2 >> 16) & 0xff] ^ cTd2[(s1 >>  8) & 0xff] ^ cTd3[s0 & 0xff] ^ rdk[15];
    /* round 4: */
    s0 = cTd0[t0 >> 24] ^ cTd1[(t3 >> 16) & 0xff] ^ cTd2[(t2 >>  8) & 0xff] ^ cTd3[t1 & 0xff] ^ rdk[16];
    s1 = cTd0[t1 >> 24] ^ cTd1[(t0 >> 16) & 0xff] ^ cTd2[(t3 >>  8) & 0xff] ^ cTd3[t2 & 0xff] ^ rdk[17];
    s2 = cTd0[t2 >> 24] ^ cTd1[(t1 >> 16) & 0xff] ^ cTd2[(t0 >>  8) & 0xff] ^ cTd3[t3 & 0xff] ^ rdk[18];
    s3 = cTd0[t3 >> 24] ^ cTd1[(t2 >> 16) & 0xff] ^ cTd2[(t1 >>  8) & 0xff] ^ cTd3[t0 & 0xff] ^ rdk[19];
    /* round 5: */
    t0 = cTd0[s0 >> 24] ^ cTd1[(s3 >> 16) & 0xff] ^ cTd2[(s2 >>  8) & 0xff] ^ cTd3[s1 & 0xff] ^ rdk[20];
    t1 = cTd0[s1 >> 24] ^ cTd1[(s0 >> 16) & 0xff] ^ cTd2[(s3 >>  8) & 0xff] ^ cTd3[s2 & 0xff] ^ rdk[21];
    t2 = cTd0[s2 >> 24] ^ cTd1[(s1 >> 16) & 0xff] ^ cTd2[(s0 >>  8) & 0xff] ^ cTd3[s3 & 0xff] ^ rdk[22];
    t3 = cTd0[s3 >> 24] ^ cTd1[(s2 >> 16) & 0xff] ^ cTd2[(s1 >>  8) & 0xff] ^ cTd3[s0 & 0xff] ^ rdk[23];
    /* round 6: */
    s0 = cTd0[t0 >> 24] ^ cTd1[(t3 >> 16) & 0xff] ^ cTd2[(t2 >>  8) & 0xff] ^ cTd3[t1 & 0xff] ^ rdk[24];
    s1 = cTd0[t1 >> 24] ^ cTd1[(t0 >> 16) & 0xff] ^ cTd2[(t3 >>  8) & 0xff] ^ cTd3[t2 & 0xff] ^ rdk[25];
    s2 = cTd0[t2 >> 24] ^ cTd1[(t1 >> 16) & 0xff] ^ cTd2[(t0 >>  8) & 0xff] ^ cTd3[t3 & 0xff] ^ rdk[26];
    s3 = cTd0[t3 >> 24] ^ cTd1[(t2 >> 16) & 0xff] ^ cTd2[(t1 >>  8) & 0xff] ^ cTd3[t0 & 0xff] ^ rdk[27];
    /* round 7: */
    t0 = cTd0[s0 >> 24] ^ cTd1[(s3 >> 16) & 0xff] ^ cTd2[(s2 >>  8) & 0xff] ^ cTd3[s1 & 0xff] ^ rdk[28];
    t1 = cTd0[s1 >> 24] ^ cTd1[(s0 >> 16) & 0xff] ^ cTd2[(s3 >>  8) & 0xff] ^ cTd3[s2 & 0xff] ^ rdk[29];
    t2 = cTd0[s2 >> 24] ^ cTd1[(s1 >> 16) & 0xff] ^ cTd2[(s0 >>  8) & 0xff] ^ cTd3[s3 & 0xff] ^ rdk[30];
    t3 = cTd0[s3 >> 24] ^ cTd1[(s2 >> 16) & 0xff] ^ cTd2[(s1 >>  8) & 0xff] ^ cTd3[s0 & 0xff] ^ rdk[31];
    /* round 8: */
    s0 = cTd0[t0 >> 24] ^ cTd1[(t3 >> 16) & 0xff] ^ cTd2[(t2 >>  8) & 0xff] ^ cTd3[t1 & 0xff] ^ rdk[32];
    s1 = cTd0[t1 >> 24] ^ cTd1[(t0 >> 16) & 0xff] ^ cTd2[(t3 >>  8) & 0xff] ^ cTd3[t2 & 0xff] ^ rdk[33];
    s2 = cTd0[t2 >> 24] ^ cTd1[(t1 >> 16) & 0xff] ^ cTd2[(t0 >>  8) & 0xff] ^ cTd3[t3 & 0xff] ^ rdk[34];
    s3 = cTd0[t3 >> 24] ^ cTd1[(t2 >> 16) & 0xff] ^ cTd2[(t1 >>  8) & 0xff] ^ cTd3[t0 & 0xff] ^ rdk[35];
    /* round 9: */
    t0 = cTd0[s0 >> 24] ^ cTd1[(s3 >> 16) & 0xff] ^ cTd2[(s2 >>  8) & 0xff] ^ cTd3[s1 & 0xff] ^ rdk[36];
    t1 = cTd0[s1 >> 24] ^ cTd1[(s0 >> 16) & 0xff] ^ cTd2[(s3 >>  8) & 0xff] ^ cTd3[s2 & 0xff] ^ rdk[37];
    t2 = cTd0[s2 >> 24] ^ cTd1[(s1 >> 16) & 0xff] ^ cTd2[(s0 >>  8) & 0xff] ^ cTd3[s3 & 0xff] ^ rdk[38];
    t3 = cTd0[s3 >> 24] ^ cTd1[(s2 >> 16) & 0xff] ^ cTd2[(s1 >>  8) & 0xff] ^ cTd3[s0 & 0xff] ^ rdk[39];
    if (Nr > 10) {
        /* round 10: */
        s0 = cTd0[t0 >> 24] ^ cTd1[(t3 >> 16) & 0xff] ^ cTd2[(t2 >>  8) & 0xff] ^ cTd3[t1 & 0xff] ^ rdk[40];
        s1 = cTd0[t1 >> 24] ^ cTd1[(t0 >> 16) & 0xff] ^ cTd2[(t3 >>  8) & 0xff] ^ cTd3[t2 & 0xff] ^ rdk[41];
        s2 = cTd0[t2 >> 24] ^ cTd1[(t1 >> 16) & 0xff] ^ cTd2[(t0 >>  8) & 0xff] ^ cTd3[t3 & 0xff] ^ rdk[42];
        s3 = cTd0[t3 >> 24] ^ cTd1[(t2 >> 16) & 0xff] ^ cTd2[(t1 >>  8) & 0xff] ^ cTd3[t0 & 0xff] ^ rdk[43];
        /* round 11: */
        t0 = cTd0[s0 >> 24] ^ cTd1[(s3 >> 16) & 0xff] ^ cTd2[(s2 >>  8) & 0xff] ^ cTd3[s1 & 0xff] ^ rdk[44];
        t1 = cTd0[s1 >> 24] ^ cTd1[(s0 >> 16) & 0xff] ^ cTd2[(s3 >>  8) & 0xff] ^ cTd3[s2 & 0xff] ^ rdk[45];
        t2 = cTd0[s2 >> 24] ^ cTd1[(s1 >> 16) & 0xff] ^ cTd2[(s0 >>  8) & 0xff] ^ cTd3[s3 & 0xff] ^ rdk[46];
        t3 = cTd0[s3 >> 24] ^ cTd1[(s2 >> 16) & 0xff] ^ cTd2[(s1 >>  8) & 0xff] ^ cTd3[s0 & 0xff] ^ rdk[47];
        if (Nr > 12) {
            /* round 12: */
            s0 = cTd0[t0 >> 24] ^ cTd1[(t3 >> 16) & 0xff] ^ cTd2[(t2 >>  8) & 0xff] ^ cTd3[t1 & 0xff] ^ rdk[48];
            s1 = cTd0[t1 >> 24] ^ cTd1[(t0 >> 16) & 0xff] ^ cTd2[(t3 >>  8) & 0xff] ^ cTd3[t2 & 0xff] ^ rdk[49];
            s2 = cTd0[t2 >> 24] ^ cTd1[(t1 >> 16) & 0xff] ^ cTd2[(t0 >>  8) & 0xff] ^ cTd3[t3 & 0xff] ^ rdk[50];
            s3 = cTd0[t3 >> 24] ^ cTd1[(t2 >> 16) & 0xff] ^ cTd2[(t1 >>  8) & 0xff] ^ cTd3[t0 & 0xff] ^ rdk[51];
            /* round 13: */
            t0 = cTd0[s0 >> 24] ^ cTd1[(s3 >> 16) & 0xff] ^ cTd2[(s2 >>  8) & 0xff] ^ cTd3[s1 & 0xff] ^ rdk[52];
            t1 = cTd0[s1 >> 24] ^ cTd1[(s0 >> 16) & 0xff] ^ cTd2[(s3 >>  8) & 0xff] ^ cTd3[s2 & 0xff] ^ rdk[53];
            t2 = cTd0[s2 >> 24] ^ cTd1[(s1 >> 16) & 0xff] ^ cTd2[(s0 >>  8) & 0xff] ^ cTd3[s3 & 0xff] ^ rdk[54];
            t3 = cTd0[s3 >> 24] ^ cTd1[(s2 >> 16) & 0xff] ^ cTd2[(s1 >>  8) & 0xff] ^ cTd3[s0 & 0xff] ^ rdk[55];
        }
    }
    rdk += Nr << 2;

    pt[offset + 0] =
        (cTd4[(t0 >> 24)       ] & 0xff000000) ^
        (cTd4[(t3 >> 16) & 0xff] & 0x00ff0000) ^
        (cTd4[(t2 >>  8) & 0xff] & 0x0000ff00) ^
        (cTd4[(t1      ) & 0xff] & 0x000000ff) ^
        rdk[0];
    pt[offset + 1] =
        (cTd4[(t1 >> 24)       ] & 0xff000000) ^
        (cTd4[(t0 >> 16) & 0xff] & 0x00ff0000) ^
        (cTd4[(t3 >>  8) & 0xff] & 0x0000ff00) ^
        (cTd4[(t2      ) & 0xff] & 0x000000ff) ^
        rdk[1];
    pt[offset + 2] =
        (cTd4[(t2 >> 24)       ] & 0xff000000) ^
        (cTd4[(t1 >> 16) & 0xff] & 0x00ff0000) ^
        (cTd4[(t0 >>  8) & 0xff] & 0x0000ff00) ^
        (cTd4[(t3      ) & 0xff] & 0x000000ff) ^
        rdk[2];
    pt[offset + 3] =
        (cTd4[(t3 >> 24)       ] & 0xff000000) ^
        (cTd4[(t2 >> 16) & 0xff] & 0x00ff0000) ^
        (cTd4[(t1 >>  8) & 0xff] & 0x0000ff00) ^
        (cTd4[(t0      ) & 0xff] & 0x000000ff) ^
        rdk[3];
}
